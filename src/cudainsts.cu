#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <errno.h>
#include <assert.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>
#include <sys/mman.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "cubar.h"

// CUDA must already have been initialized before calling cudaid().
#define CUDASTRLEN 80
static int
id_cuda(int dev,unsigned *mem,unsigned *tmem,int *state){
	struct hipDeviceProp_t dprop;
	int major,minor,attr,cerr;
	void *str = NULL;
	hipCtx_t ctx;
	hipDevice_t c;

	*state = 0;
	if((cerr = hipDeviceGet(&c,dev)) != hipSuccess){
		fprintf(stderr," Couldn't associative with device (%d)\n",cerr);
		return -1;
	}
	if((cerr = hipGetDeviceProperties(&dprop,dev)) != hipSuccess){
		fprintf(stderr," Couldn't get device properties (%d)\n",cerr);
		return -1;
	}
	cerr = hipDeviceGetAttribute(&attr,hipDeviceAttributeMultiprocessorCount,c);
	if(cerr != hipSuccess || attr <= 0){
		return -1;
	}
	if((cerr = hipDeviceComputeCapability(&major,&minor,c)) != hipSuccess){
		return -1;
	}
	if((str = malloc(CUDASTRLEN)) == NULL){
		return -1;
	}
	if((cerr = hipDeviceGetName((char *)str,CUDASTRLEN,c)) != hipSuccess){
		goto err;
	}
	if((cerr = hipCtxCreate(&ctx,hipDeviceMapHost|hipDeviceScheduleYield,c)) != hipSuccess){
		fprintf(stderr," Couldn't create context (%d)\n",cerr);
		goto err;
	}
	size_t cudatmem,cudamem;
	if((cerr = hipMemGetInfo(&cudamem,&cudatmem)) != hipSuccess){
		cuCtxDetach(ctx);
		goto err;
	}
	*mem = cudamem;
	*tmem = cudatmem;
	*state = dprop.computeMode;
	if(printf("%d.%d %s %s %u/%uMB free %s\n",
		major,minor,
		dprop.integrated ? "Integrated" : "Standalone",(char *)str,
		*mem / (1024 * 1024) + !!(*mem / (1024 * 1024)),
		*tmem / (1024 * 1024) + !!(*tmem / (1024 * 1024)),
		*state == hipComputeModeExclusive ? "(exclusive)" :
		*state == hipComputeModeProhibited ? "(prohibited)" :
		*state == hipComputeModeDefault ? "(shared)" :
		"(unknown compute mode)") < 0){
		cerr = -1;
		goto err;
	}
	free(str);
	return hipSuccess;

err:	// cerr ought already be set!
	free(str);
	return cerr;
}

#define GIDX ((gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * blockIdx.y + \
		 blockIdx.x)

#define BIDX ((blockDim.x * blockDim.y) * threadIdx.z + blockDim.x * threadIdx.y + \
		 threadIdx.x)

#define ABSIDX (((GIDX) * blockDim.x * blockDim.y * blockDim.z) + BIDX)

__global__ void memkernel(uint64_t *t0,uint64_t *t1){
	t0[ABSIDX] = clock64();
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void shlkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "shl.b32 %0, %1, %2;" : "=r"(pa) : "r"(pb), "r"(pc) );
		asm( "shl.b32 %0, %1, %2;" : "=r"(pa1) : "r"(pb1), "r"(pc1) );
		asm( "shl.b32 %0, %1, %2;" : "=r"(pb) : "r"(pc), "r"(pa) );
		asm( "shl.b32 %0, %1, %2;" : "=r"(pb1) : "r"(pc1), "r"(pa1) );
		asm( "shl.b32 %0, %1, %2;" : "=r"(pc) : "r"(pa), "r"(pb) );
		asm( "shl.b32 %0, %1, %2;" : "=r"(pc1) : "r"(pa1), "r"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void shrkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "shr.b32 %0, %1, %2;" : "=r"(pa) : "r"(pb), "r"(pc) );
		asm( "shr.b32 %0, %1, %2;" : "=r"(pa1) : "r"(pb1), "r"(pc1) );
		asm( "shr.b32 %0, %1, %2;" : "=r"(pb) : "r"(pc), "r"(pa) );
		asm( "shr.b32 %0, %1, %2;" : "=r"(pb1) : "r"(pc1), "r"(pa1) );
		asm( "shr.b32 %0, %1, %2;" : "=r"(pc) : "r"(pa), "r"(pb) );
		asm( "shr.b32 %0, %1, %2;" : "=r"(pc1) : "r"(pa1), "r"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void faddkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	double pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "add.f64 %0, %1, %2;" : "=d"(pa) : "d"(pb), "d"(pc) );
		asm( "add.f64 %0, %1, %2;" : "=d"(pa1) : "d"(pb1), "d"(pc1) );
		asm( "add.f64 %0, %1, %2;" : "=d"(pb) : "d"(pc), "d"(pa) );
		asm( "add.f64 %0, %1, %2;" : "=d"(pb1) : "d"(pc1), "d"(pa1) );
		asm( "add.f64 %0, %1, %2;" : "=d"(pc) : "d"(pa), "d"(pb) );
		asm( "add.f64 %0, %1, %2;" : "=d"(pc1) : "d"(pa1), "d"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void addkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "add.u32 %0, %1, %2;" : "=r"(pa) : "r"(pb), "r"(pc) );
		asm( "add.u32 %0, %1, %2;" : "=r"(pa1) : "r"(pb1), "r"(pc1) );
		asm( "add.u32 %0, %1, %2;" : "=r"(pb) : "r"(pc), "r"(pa) );
		asm( "add.u32 %0, %1, %2;" : "=r"(pb1) : "r"(pc1), "r"(pa1) );
		asm( "add.u32 %0, %1, %2;" : "=r"(pc) : "r"(pa), "r"(pb) );
		asm( "add.u32 %0, %1, %2;" : "=r"(pc1) : "r"(pa1), "r"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void add64kernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	uint64_t pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "add.u64 %0, %1, %2;" : "=l"(pa) : "l"(pb), "l"(pc) );
		asm( "add.u64 %0, %1, %2;" : "=l"(pa1) : "l"(pb1), "l"(pc1) );
		asm( "add.u64 %0, %1, %2;" : "=l"(pb) : "l"(pc), "l"(pa) );
		asm( "add.u64 %0, %1, %2;" : "=l"(pb1) : "l"(pc1), "l"(pa1) );
		asm( "add.u64 %0, %1, %2;" : "=l"(pc) : "l"(pa), "l"(pb) );
		asm( "add.u64 %0, %1, %2;" : "=l"(pc1) : "l"(pa1), "l"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void mulkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;


	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pa) : "r"(pb), "r"(pc) );
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pa1) : "r"(pb1), "r"(pc1) );
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pb) : "r"(pc), "r"(pa) );
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pb1) : "r"(pc1), "r"(pa1) );
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pc) : "r"(pa), "r"(pb) );
		asm( "mul.lo.u32 %0, %1, %2;" : "=r"(pc1) : "r"(pa1), "r"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void vaddr3kernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2;
	unsigned z;

	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pa) : "r"(pb), "r"(pc) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pa1) : "r"(pb1), "r"(pc1) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pb) : "r"(pc), "r"(pa) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pb1) : "r"(pc1), "r"(pa1) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pc) : "r"(pa), "r"(pb) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %2;" : "=r"(pc1) : "r"(pa1), "r"(pb1) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

__global__ void vaddkernel(uint64_t *t0,uint64_t *t1,const unsigned loops){
	unsigned pa,pb = 1,pc = 2,pa1,pb1 = 1,pc1 = 2,pd = 3,pd1 = 3;
	unsigned z;

	t0[ABSIDX] = clock64();
#pragma unroll 16
	for(z = 0 ; z < loops ; ++z){
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pa) : "r"(pb), "r"(pc), "r"(pd) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pa1) : "r"(pb1), "r"(pc1), "r"(pd1) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pb) : "r"(pc), "r"(pa), "r"(pd) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pb1) : "r"(pc1), "r"(pa1), "r"(pd1) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pc) : "r"(pa), "r"(pb), "r"(pd) );
		asm( "vadd.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(pc1) : "r"(pa1), "r"(pb1), "r"(pd) );
	}
	t1[ABSIDX] = pc1 + pc;
	t0[ABSIDX] = clock64() - t0[ABSIDX];
}

static void
stats(const struct timeval *tv0,const struct timeval *tv1,
		const uint64_t *t0,const uint64_t *t1,unsigned n,
		unsigned loops){
	uintmax_t sumdelt = 0;
	struct timeval tv;
	uint64_t res;
	unsigned z;

	res = *t1;
	timersub(tv1,tv0,&tv);
	printf("\tKernel wall time: %ld.%06lds\n",tv.tv_sec,tv.tv_usec);
	for(z = 0 ; z < n ; ++z){
		//printf("delt: %lu res: %u\n",t0[z],t1[z]);
		sumdelt += t0[z];
		assert(res == t1[z]);
	}
	printf("\tMean cycles / thread: %ju cycles / op: %ju\n",sumdelt / n,sumdelt / n / loops);
}

static int
check_const_ram(const unsigned loops){
	dim3 dblock(BLOCK_SIZE,1,1);
	struct timeval tv0, tv1;
	dim3 dgrid(1,1,1);
	uint64_t *h0,*h1;
	uint64_t *t0,*t1;
	size_t s;

	s = (dgrid.x * dgrid.y * dgrid.z) * (dblock.x * dblock.y * dblock.z);
	h0 = new uint64_t[s];
	h1 = new uint64_t[s];
	if(hipMalloc(&t0,s * sizeof(*t0)) != hipSuccess){
		fprintf(stderr,"\n  Error allocating %zu t0 bytes\n",s);
		free(h1); free(h0);
		return -1;
	}
	if(hipMalloc(&t1,s * sizeof(*t0)) != hipSuccess){
		fprintf(stderr,"\n  Error allocating %zu t0 bytes\n",s);
		hipFree(t0); free(h1); free(h0);
		return -1;
	}

	printf("Timing 64-bit store+load+store...");
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	memkernel<<<dblock,dgrid>>>(t0,t1);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,1);

	printf("Timing %u adds...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	addkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u 64-bit adds...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	add64kernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u 64-bit floating-point adds...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	faddkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u muls...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	mulkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u vadds...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	vaddkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u vadds (duplicated registers)...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	vaddr3kernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u shls...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	shlkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	printf("Timing %u shrs...",loops);
	fflush(stdout);
	gettimeofday(&tv0,NULL);
	shrkernel<<<dblock,dgrid>>>(t0,t1,loops);
	if(hipCtxSynchronize() ||
			hipMemcpy(h0,t0,s * sizeof(*h0),hipMemcpyDeviceToHost) != hipSuccess ||
			hipMemcpy(h1,t1,s * sizeof(*h1),hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err;

		err = hipGetLastError();
		fprintf(stderr,"\n  Error timing instruction (%s?)\n",
				hipGetErrorString(err));
		goto err;
	}
	gettimeofday(&tv1,NULL);
	printf("good.\n");
	stats(&tv0,&tv1,h0,h1,s,loops * 6);

	hipFree(t1); hipFree(t0);
	free(h1); free(h0);
	return 0;

err:
	hipFree(t1); hipFree(t0);
	free(h1); free(h0);
	return -1;
}

#define LOOPS (0x00010000u)

static void
usage(const char *a0,int status){
	fprintf(stderr,"usage: %s [loops]\n",a0);
	fprintf(stderr," default loopcount: %u\n",LOOPS);
	exit(status);
}

int main(int argc,char **argv){
	unsigned long loops;
	int z,count;

	if(argc > 2){
		usage(argv[0],EXIT_FAILURE);
	}else if(argc == 2){
		if(getzul(argv[1],&loops)){
			usage(argv[0],EXIT_FAILURE);
		}
	}else{
		loops = LOOPS;
	}
	if(init_cuda_alldevs(&count)){
		return EXIT_FAILURE;
	}
	printf("CUDA device count: %d\n",count);
	for(z = 0 ; z < count ; ++z){
		uint64_t hostresarr[GRID_SIZE * BLOCK_SIZE];
		unsigned mem,tmem;
		uint64_t *resarr;
		int state;

		printf(" %03d ",z);
		if(id_cuda(z,&mem,&tmem,&state)){
			return EXIT_FAILURE;
		}
		if(hipDeviceSetCacheConfig(hipFuncCachePreferL1) != hipSuccess){
			fprintf(stderr,"Error preferring L1 to shared memory.\n");
		}
		if(state != hipComputeModeDefault){
			printf("  Skipping device %d (put it in shared mode).\n",z);
			continue;
		}
		if(hipMalloc(&resarr,sizeof(hostresarr)) || hipMemset(resarr,0,sizeof(hostresarr))){
			fprintf(stderr," Couldn't allocate result array (%s?)\n",
				hipGetErrorString(hipGetLastError()));
			return EXIT_FAILURE;
		}
		if(check_const_ram(loops)){
			return EXIT_FAILURE;
		}
		printf(" Success.\n");
		if(hipMemcpy(hostresarr,resarr,sizeof(hostresarr),hipMemcpyDeviceToHost) || hipFree(resarr)){
			fprintf(stderr," Couldn't free result array (%s?)\n",
				hipGetErrorString(hipGetLastError()));
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
