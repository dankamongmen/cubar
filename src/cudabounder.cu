#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda8803ss.h"

static void
usage(const char *a0){
	fprintf(stderr,"usage: %s devno\n",a0);
}

static int
basic_params(hipDeviceptr_t p,size_t s){
	hipDeviceptr_t p2;
	hipError_t cerr;

	if( (cerr = hipMalloc(&p2,s)) || (cerr = hipMemsetD8(p2,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		return -1;
	}
	printf("Got secondary %zub allocation at %p\n",s,p2);
	if( (cerr = hipFree(p2)) ){
		fprintf(stderr,"Couldn't free %zu base (%d)\n",s,cerr);
		return -1;
	}
	// FIXME not very rigorous, not at all...[frown]
	printf("Minimum cuMalloc() alignment might be: %u\n",p2 - p);
	return 0;
}

#define BYTES_PER_KERNEL 4

__global__ void
touchbytes(hipDeviceptr_t ptr,uint32_t off,hipDeviceptr_t res){
	uint8_t b;

	b = *(unsigned char *)((uintptr_t)ptr + off + blockIdx.x);
	if(b == 0xff){
		*(uint32_t *)((uintptr_t)res + blockIdx.x * BYTES_PER_KERNEL) = 1;
	}
}

static int
shoveover(hipDeviceptr_t *r,size_t s){
	const size_t shovelen = 0xf00000;
	hipDeviceptr_t tmp;
	hipError_t cerr;

	if( (cerr = hipMalloc(&tmp,shovelen)) ){
		fprintf(stderr,"Couldn't alloc+init %zu shove (%d)\n",shovelen,cerr);
		return -1;
	}
	printf("Got %zub shovebuf at %p\n",shovelen,tmp);
	if( (cerr = hipMalloc(r,s)) || (cerr = hipMemsetD32(*r,0,s / sizeof(uint32_t))) ){
		fprintf(stderr,"Couldn't alloc+init %zu resarr (%d)\n",s,cerr);
		return -1;
	}
	if( (cerr = hipFree(tmp)) ){
		fprintf(stderr,"Couldn't free %zu shove (%d)\n",shovelen,cerr);
		return -1;
	}
	return 0;
}

int main(int argc,char **argv){
	hipDeviceptr_t ptr,res;
	unsigned long zul;
	hipCtx_t ctx;
	hipError_t cerr;
	size_t s,z;

	if(argc != 2 || getzul(argv[1],&zul)){
		usage(*argv);
		exit(EXIT_FAILURE);
	}
	if(init_cuda_ctx(zul,&ctx)){
		exit(EXIT_FAILURE);
	}
	s = sizeof(ptr);
	if( (cerr = hipMalloc(&ptr,s)) || (cerr = hipMemsetD8(ptr,0xff,s)) ){
		fprintf(stderr,"Couldn't alloc+init %zu base (%d)\n",s,cerr);
		exit(EXIT_FAILURE);
	}
	printf("Got base %zub allocation at %p\n",s,ptr);
	if(basic_params(ptr,s)){
		exit(EXIT_FAILURE);
	}
	if(shoveover(&res,BYTES_PER_KERNEL * sizeof(uint32_t))){
		exit(EXIT_FAILURE);
	}
	if(res <= ptr){ // FIXME...see loop detect below
		fprintf(stderr,"Unexpected pointer arrangement (%p >= %p)\n",ptr,res);
		exit(EXIT_FAILURE);
	}
	printf("Got %zub resarr at %p (%ub gap)\n",
			BYTES_PER_KERNEL * sizeof(uint32_t),res,res - ptr);
	z = 0;
	while((cerr = hipCtxSynchronize()) == hipSuccess){
		dim3 dg(1,1,1),db(BYTES_PER_KERNEL,1,1);

		touchbytes<<<dg,db>>>(ptr,z,res);
		// FIXME check res
		if(((z += BYTES_PER_KERNEL) + ptr) > res){
			printf("Hit result array at %p; breaking loop\n",res);
			break;
		}
	}
	printf("Exited loop (ret: %d) at %zu\n",cerr,z);
	exit(EXIT_SUCCESS);
}
