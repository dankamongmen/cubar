#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  include <windows.h>
#endif
#include <cuda_gl_interop.h>
#include "../../common/inc/cutil.h"

extern "C" void process(int pbo_in, int pbo_out, int width, int height, int radius);
extern "C" void pboRegister(int pbo);
extern "C" void pboUnregister(int pbo);

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ int getPixel(int *data, int x, int y, int width, int height)
{
    x = clamp(x, 0, width-1);
    y = clamp(y, 0, height-1);
    return data[y*width+x];
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

__global__ void
cudaProcess(int* g_data, int* g_odata, int imgw, int imgh, int tilew, int r, float threshold, float highlight)
{
    extern __shared__ int sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(g_data, x, y, imgw, imgh);

    // borders
    if (threadIdx.x < r) {
        // left
        SMEM(tx, r + ty) = getPixel(g_data, x - r, y, imgw, imgh);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(g_data, x + bw, y, imgw, imgh);
    }
    if (threadIdx.y < r) {
        // top
        SMEM(r + tx, ty) = getPixel(g_data, x, y - r, imgw, imgh);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(g_data, x, y + bh, imgw, imgh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r)) {
        // tl
        SMEM(tx, ty) = getPixel(g_data, x - r, y - r, imgw, imgh);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(g_data, x - r, y + bh, imgw, imgh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(g_data, x + bh, y - r, imgw, imgh);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(g_data, x + bw, y + bh, imgw, imgh);
    }

    // wait for loads to complete
    __syncthreads();

    // perform convolution
    float rsum = 0.0;
    float gsum = 0.0;
    float bsum = 0.0;
    float samples = 0.0;

    for(int dy=-r; dy<=r; dy++) {
        for(int dx=-r; dx<=r; dx++) {
#if 0
            // try this to see the benefit of using shared memory
            int pixel = getPixel(g_data, x+dx, y+dy, imgw, imgh);
#else
            int pixel = SMEM(r+tx+dx, r+ty+dy);
#endif

            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;
            if (l <= r*r) {
                float r = float(pixel&0xff);
                float g = float((pixel>>8)&0xff);
                float b = float((pixel>>16)&0xff);
#if 1
                // brighten highlights
                float lum = (r + g + b) / (255*3);
                if (lum > threshold) {
                    r *= highlight;
                    g *= highlight;
                    b *= highlight;
                }
#endif
                rsum += r;
                gsum += g;
                bsum += b;
                samples += 1.0;
            }
        }
    }

    rsum /= samples;
    gsum /= samples;
    bsum /= samples;
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void process( int pbo_in, int pbo_out, int width, int height, int radius) 
{
    int *in_data;
    int* out_data;

    CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&in_data, pbo_in));
    CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&out_data, pbo_out));

    dim3 block(16, 16, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    int sbytes = (block.x+(2*radius))*(block.y+(2*radius))*sizeof(int);

    cudaProcess<<< grid, block, sbytes>>>(in_data, out_data, width, height, block.x+(2*radius), radius, 0.8, 4.0);

    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( pbo_in));
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( pbo_out));
}

void pboRegister(int pbo)
{
    // register this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
}
void pboUnregister(int pbo)
{
    // unregister this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));
}
